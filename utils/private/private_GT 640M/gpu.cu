
#include <hip/hip_runtime.h>
/*	>> !nvcc -arch=sm_21 -ptx gpu.cu

	The format of gpu.ptx may require a workaround: each global function is defined by
	a line starting with ".entry" followed by the mangled function name. MATLABs just in
	time compiler requires a space between the function name and the argument bracket.
*/


/*	Accumulate partial products from an image in the terms.

	    No boundary and consistency check is performed!
	    For argument format checking, call cpu instead.

	terms=cpu(terms,image,tasks);

	tasks	[pixels products factors(1) factor{1} factors(2) factor{2} ...]

	    pixels	Image size X*Y*Z
	    products	Number of terms
	    factors	Number of factors
	    factor	Pixels of factors (ascending)

	terms(x,y,t) += prod(image(x+y*X+factor{t})

	    x,y		Coordinates
	    t		Product term
*/
__global__ void gpu(double* terms, const double* image, const int* tasks)
{	int pixel=threadIdx.x + blockDim.x*blockIdx.x;
	int pixels=tasks[0];				// number of pixels
	if(pixel < pixels)
	{	int task=tasks[1];			// number of products
		tasks+=2;
		terms+=pixel;				// pixel (x,y)
		image+=pixel;
		pixel=pixels - pixel;			// remaining pixels
		while(--task >= 0)
		{	int factor=*tasks;		// number of factors
			if (tasks[factor] < pixel)
			{	double term=image[tasks[factor]];
				while(--factor > 0) term*=image[tasks[factor]];
				*terms+=term;
			}
			tasks+=1+*tasks;
			terms+=pixels;
		}
	}
}
